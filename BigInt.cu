#include "hip/hip_runtime.h"
#include "BigInt.h"

#include <iostream>

// конструткор по умолчанию
CUDA_MEMBER BigInt::BigInt() {
	this->isNeg = false;
	this->value = "0";
}

// конструктор из стандартного целого числа
CUDA_MEMBER BigInt::BigInt(long x) {
	this->isNeg = x < 0;
	this->value = std::to_string(isNeg ? -x : x);
}

// конструктор из строки (пустая строка создаст число 0)
CUDA_MEMBER BigInt::BigInt(const std::string &value) {
	if (!value.length()) {
		this->value = "0";
		isNeg = false;

		return;
	}		

	isNeg = value[0] == '-';
	this->value = value.substr(isNeg);

	// определяем число ведущих нулей в строке
	int count = 0;
	while (this->value[count] == '0' && this->value.length() - count > 1)
		count++;

	this->value = this->value.substr(count); // удаляем ведущие нули

	// проверяем "на цифру" каждый символ строки, кидаем исключение, если есть другие символы
	for(size_t i = 0; i < this->value.length(); i++)
		if (this->value[i] < '0' || this->value[i] > '9')
			throw std::string("BigInt(const string &value) - string contain incorrect characters: ") + this->value;
}

// конструктор копирования
CUDA_MEMBER BigInt::BigInt(const BigInt& bigInt) {
	this->value = bigInt.getValue();
	this->isNeg = bigInt.getIsNeg();
}

// получение строки со значением числа
CUDA_MEMBER const std::string &BigInt::getValue() const {
	return value;
}

// получение флага отрицательности числа
CUDA_MEMBER const bool BigInt::getIsNeg() const {
	return isNeg;
}

// изменение флага отрицательности числа
CUDA_MEMBER void BigInt::setIsNeg(bool isNeg) {
	this->isNeg = isNeg;
}

// получение знака числа
CUDA_MEMBER int BigInt::sign() const {
	return (1 - 2 * isNeg) * (value != "0");
}

// проверка на чётность
CUDA_MEMBER const bool BigInt::isEven() const {
	return (value[value.length() - 1] - '0') % 2 == 0;
}

// получение модуля числа
CUDA_MEMBER BigInt BigInt::abs() const {
	return BigInt(value);
}

// получение числа в степени n
CUDA_MEMBER BigInt BigInt::pow(long n) const {
	if (!n)
		return 1;

	if (n & 1)
		return pow(n - 1) * *this;
	else {
		BigInt tmp = pow(n / 2);
		return tmp * tmp;
	}
}

// вычисление корня n-ой степени из числа
CUDA_MEMBER BigInt BigInt::sqrt(long n) const {
	if (n < 2)
		throw std::string("BigInt::sqrt() - n must be 2 or more");

	if (isNeg && ((n & 1)== 0))
		throw std::string("BigInt::sqrt() - value must be positive");

	size_t length = (value.length() + 1) / 2;
	size_t index = 0;

	std::string v(length, '0');

	while (index < length) {
		v[index] = '9';

		while (BigInt(v).pow(n) > this->abs() && v[index] > '0')
			v[index]--;

		index++;
	}

	return isNeg ? std::string("-") + v : v;
}

// проверка на равенство двух чисел (равны, если одного знака и одного значения)
CUDA_MEMBER const bool BigInt::operator==(const BigInt &bigInt) const {
	return (value == bigInt.getValue()) && (isNeg == bigInt.getIsNeg());
}

// проверка на неравенство - отрицание равенства
CUDA_MEMBER const bool BigInt::operator!=(const BigInt &bigInt) const {
	return !(*this == bigInt);
}

// проверка, что число меньше bigInt
CUDA_MEMBER const bool BigInt::operator<(const BigInt &bigInt) const {
	std::string value2 = bigInt.getValue(); // получаем значение второго числа
	size_t len1 = value.length(); // запоминаем длину первого числа
	size_t len2 = value2.length(); // запоминаем длину второго числа

	// если знаки одинаковые, то проверяем значения
	if (isNeg == bigInt.getIsNeg()) {
		// если длины не равны
		if (len1 != len2)
			return (len1 < len2) ^ isNeg; // меньше число с меньшей длинной для положительных и с большей длиной для отрицательных

		size_t i = 0;

		// ищем разряд, в котором значения отличаются
		while (i < len1 && value[i] == value2[i])
			i++;

		// если разряд найден, то меньше число с меньшей цифрой для положительных и с большей цифрой для отрицательных, иначе числа равны
		return (i < len1) && ((value[i] < value2[i]) ^ isNeg);
	}

	return isNeg; // знаки разные, если число отрицательное, то оно меньше, если положительное, то больше
}

// проверка, что число больше bigInt
CUDA_MEMBER const bool BigInt::operator>(const BigInt &bigInt) const {
	return !(*this < bigInt || *this == bigInt);
}

// проверка, что число меньше или равно bigInt
CUDA_MEMBER const bool BigInt::operator<=(const BigInt &bigInt) const {
	return *this < bigInt || *this == bigInt;
}

// проверка, что число больше или равно bigInt
CUDA_MEMBER const bool BigInt::operator>=(const BigInt &bigInt) const {
	return *this > bigInt || *this == bigInt;
}

// операция присваивания
CUDA_MEMBER BigInt &BigInt::operator=(const BigInt &bigInt) {
	value = bigInt.getValue();
	isNeg = bigInt.getIsNeg();

	return *this;
}

// унарный минус, если было отрицательным, возвращаем положительное, иначе отрицательное
CUDA_MEMBER BigInt BigInt::operator-() const && {
	return BigInt(isNeg ? value : std::string("-") + value);
}

// унарный плюс (просто копируем значение числа)
CUDA_MEMBER BigInt BigInt::operator+() const && {
	return BigInt(*this);
}

// умножение методом Карацубы
CUDA_MEMBER BigInt BigInt::karatsuba_mul(const BigInt &a, const BigInt &b) {
    std::string v1 = a.getValue();
    std::string v2 = b.getValue();

    size_t len1 = v1.length();
    size_t len2 = v2.length();
    size_t len = std::max(len1, len2);

    if (len1 + len2 < 9)
        return stol(a.getValue()) * stol(b.getValue());

    len += len % 2;
    size_t n = len / 2;

    BigInt Xr(len1 > n ? v1.substr(len1 - n, n) : v1);
    BigInt Xl(a >> n);
    BigInt Yr(len2 > n ? v2.substr(len2 - n, n) : v2);
    BigInt Yl(b >> n);

    BigInt P1 = karatsuba_mul(Xl, Yl);
    BigInt P2 = karatsuba_mul(Xr, Yr); 
    BigInt P3 = karatsuba_mul(Xl + Xr, Yl + Yr);

    return (P1 << len) + ((P3 - P2 - P1) << n) + P2;
}


// бинарный плюс (сложение двух чисел)
CUDA_MEMBER BigInt BigInt::operator+(const BigInt &bigInt) const {
	bool isAddOp = !(bigInt.getIsNeg() ^ isNeg); // если знаки одинаковые, то выполняем сложение

	if (isAddOp) {
		std::string num2 = bigInt.getValue(); // запоминаем значение второго числа

		size_t len1 = value.length(); // запоминаем длину первого числа
		size_t len2 = num2.length(); // запоминаем длину второго числа
		size_t length = 1 + std::max(len1, len2);  // длина суммы равна максимуму из двух длин + 1 из-за возможного переноса разряда

		char res[length + 1]; // строковый массив для выполнения операции сложения

		res[length - 1] = res[length] = '\0';

		for (size_t i = 0; i < length - 1; i++) {
			int j = length - 1 - i;
			res[j] += ((i < len2) ? (num2[len2 - 1 - i] - '0') : 0) + ((i < len1) ? (value[len1 - 1 - i] - '0') : 0); // выполняем сложение разрядов
	    	res[j - 1] = res[j] / 10; // выполняем перенос в следущий разряд, если он был
		    res[j] = res[j] % 10 + '0'; // оставляем только единицы от возможного переноса и превращаем символ в цифру
		}

		res[0] += '0';
		
		return BigInt(isNeg ? std::string("-") + std::string(res) : std::string(res)); // возвращаем результат, в зависимости от знака`
	}
	else
		return isNeg ? (bigInt - (-BigInt(*this))) : (*this - (-BigInt(bigInt))); // одно из чисел отрицательное, а другое положительное, отправляем на вычитание, меняя знак
}

// бинарный минус (вычитание двух чисел)
CUDA_MEMBER BigInt BigInt::operator-(const BigInt &bigInt) const {
	if (*this == bigInt) 
		return 0; // если числа равны, то какой смысл вычитать?

	// если оба числа положительные, то выполняем вычитание
	if (!isNeg && !bigInt.getIsNeg()) {
		std::string value2 = bigInt.getValue(); // запоминаем значение второго числа
		
		size_t len1 = value.length(); // запоминаем длину первого числа
		size_t len2 = value2.length(); // запоминаем длину второго числа
		size_t length = std::max(len1, len2); // длина результата не превысит максимума длин чисел
		
		bool isNegRes = bigInt > *this; // определяем знак результата

		int a[length], b[length]; // массивы аргументов
		a[0] = b[0] = 0; // обнуляем нулевые элементы массивов

		char res[length + 1]; // строковый массив для результата
		res[length - 1] = res[length] = '\0'; // устанавливаем символ окончания строки

		int sign = (2 * isNegRes - 1); // получаем числовое значение знака результата

		for (size_t i = 0; i < length - 1; i++) {
			a[i] += (i < len1) ? (value[len1 - 1 - i] - '0') : 0; // формируем разряды
			b[i] += (i < len2) ? (value2[len2 - 1 - i] - '0') : 0; // из строк аргументов

			b[i + 1] = -isNegRes; // в зависимости от знака занимаем или не занимаем
			a[i + 1] = isNegRes- 1; // 10 у следующего разряда

			res[length - 1 - i] += 10 + sign * (b[i] - a[i]);
			res[length - 1 - i - 1]  = res[length - 1 - i] / 10;
			res[length - 1 - i] = res[length - 1 - i] % 10 + '0';
		}

		// выполняем операцию с последним разрядом
		a[length - 1] += (length - 1 < len1) * (value[0] - '0');
		b[length - 1] += (length - 1 < len2) * (value2[0] - '0');

		// записываем в строку последний разряд
		res[0] += sign * (b[length - 1] - a[length - 1]) + '0';

		return BigInt(isNegRes ? std::string("-") + std::string(res) : std::string(res)); // возвращаем результат, учитывая знак
	} else // если оба числа отрицательные, то меняем местами, меняем знаки и повторяем вычитание, а если знаки разные, то отправляем на сумму
		return isNeg && bigInt.getIsNeg() ? (-BigInt(bigInt) - (-BigInt(*this))) : (*this + -BigInt(bigInt)); 
}

// бинарная звёздочка (умножение двух чисел)
CUDA_MEMBER BigInt BigInt::operator*(const BigInt &bigInt) const {
	if (value == "0" || bigInt.getValue() == "0")
		return 0; // если один из множителей равен нулю, то результат равен нулю

	std::string value2 = bigInt.getValue(); // запоминаем значение второго числа
		
	size_t len1 = value.length(); // запоминаем длину первого числа
	size_t len2 = value2.length(); // запоминаем длину второго числа
	size_t length = len1 + len2 + 1; // резульат влезет в сумму длин + 1 из-за возможного переноса
	bool isNegRes = isNeg ^ bigInt.getIsNeg(); // флаг отрицательности результата - отрицательный, если числа разных знаков

	if (length < 10) { // умножаем как обычный long
		long res = stol(value) * stol(value2);
		return BigInt(isNegRes ? -res : res);
	}
	else if (length < 2000) { // умножаем в столбик
		int a[length], b[length]; // массивы аргументов и результата

		char res[length + 1]; // строковый массив для результата
		res[length] = '\0'; // устанавливаем символ окончания строки

		// заполняем массивы инверсной записью чисел (с ведущими нулями)
		for (size_t i = 0; i < length; i++) {
			a[i] = (i < len1) ? (value[len1 - 1 - i] - '0') : 0;
			b[i] = (i < len2) ? (value2[len2 - 1 - i] - '0') : 0;
			res[i] = 0;
		}

		// выполняем умножение "в столбик""
		for (size_t i = 0; i < len1; i++) {
		    for (size_t j = 0; j < len2; j++) {
		        res[length - 1 - (i + j)] += a[i] * b[j];
		        res[length - 1 - (i + j + 1)] += res[length - 1 - (i + j)] / 10;
			    res[length - 1 - (i + j)] %= 10;
		    }
		}

		// переписываем результат в строку
		for (size_t i = 0; i < length; i++)
			res[length - 1 - i] += '0';

		return BigInt(isNegRes ? std::string("-") + std::string(res) : std::string(res)); // возвращаем результат, учитывая его знак
	} else { // умножаем по методу Карацубы
		BigInt res = karatsuba_mul(*this, bigInt);
		return isNegRes ? -BigInt(res) : res;
	}
}

// бинарный слеш (деление двух чисел)
CUDA_MEMBER BigInt BigInt::operator/(const BigInt &bigInt) const {
	std::string value1 = value;
	std::string value2 = bigInt.getValue(); // запоминаем значение второго числа

	if (value2[0] == '0')
		throw std::string("Division by zero!"); // нельзя делить на ноль, этому ещё в школе учат

	if (value[0] == '0')
		return 0; // а вот ноль делить можно на всё, кроме нуля, но смысл?

	if (value2 == "1")
		return BigInt(bigInt.getIsNeg() ? -BigInt(*this) : *this); // делить на 1 можно, но смысл?

	size_t zeroes = 0;
	while (value2[value2.length() - 1 - zeroes] == '0')
		zeroes++;

	if (zeroes >= value.length())
		return 0;

	// избавляемся от круглых чисел
	if (zeroes) {
		value1 = value1.substr(0, value1.length() - zeroes);
		value2 = value2.substr(0, value2.length() - zeroes);
	}

	bool isNegRes = isNeg ^ bigInt.getIsNeg(); // считаем знак числа

	BigInt tmp(value2);

	size_t divider_length = value2.length(); // запоминаем длину делителя
	long divider_v = divider_length > 8 ? 0 : atol(value2.c_str()); // если длина больше 8, то обнуляем long'овый делитель, иначе переводим строку в long

	size_t length = value1.length(); // получаем длину делимого
	size_t index = 0; // стартуем с нулевого индекса

	std::string div; // строка результата деления
	std::string v; // строка подчисла (которое делится на делитель в столбик)

	// формируем начальное число для деления
	while (BigInt(v) < tmp && index < length)
		v += value1[index++];

	do {
		int count = 0; // результат деления подчисла на делитель

		// если можем разделить, то делим
		if (BigInt(v) >= tmp) {
			if (divider_length > 8) { // если не входит в long, то делим с помощью вычитания
				BigInt mod = v;

				while (mod >= tmp) {
					mod -= tmp;
					count++;
				}

				v = mod.getValue();
			} else {
				long mod = stol(v);
				count = mod / divider_v;
				v = std::to_string(mod % divider_v);
			}
		}
		
		div += count + '0'; // если не делили, то добавили ноль к результату, иначе добавили результат дедения

		if (index <= length)
			v += value1[index++]; // формируем новое значение для подчисла
	} while (index <= length);

	return BigInt(isNegRes && div != "0" ? std::string("-") + div : div); // возвращаем результат учитывая знак и возможное равенство нулю
}

// бинарный процент (операция взятия остатка от деления) (полностью аналогична делению)
CUDA_MEMBER BigInt BigInt::operator%(const BigInt &bigInt) const {
	std::string value2 = bigInt.getValue();

	if (value2[0] == '0')
		throw std::string("Division by zero!");

	if (value[0] == '0' || value2 == "1")
		return 0;

	if (value.length() < 9 && value2.length() < 9) {
		long res = stol(value) % stol(value2);
		return isNeg ? -res : res;
	}

	BigInt tmp(value2);

	size_t divider_length = value2.length(); // запоминаем длину делителя
	long divider_v = divider_length >= 9 ? 0 : atol(value2.c_str()); // если длина больше 8, то обнуляем long'овый делитель, иначе переводим строку в long

	size_t length = value.length();
	size_t index = 0;
	BigInt mod2 = value;
	std::string v;

	while (BigInt(v) < tmp && index < length)
		v += value[index++];

	do {
		if (BigInt(v) >= tmp) {
			if (divider_v)
				v = std::to_string(stol(v) % divider_v);
			else {
				BigInt mod = v;

				while (mod >= tmp)
					mod -= tmp;

				v = mod.getValue();
			}
		}

		if (index <= length) {
			mod2 = v;
			v += value[index++];				
		}
	} while (index <= length);

	if (mod2.getValue() == "0")
		return 0;

	return isNeg ? -BigInt(mod2) : mod2;
}

CUDA_MEMBER BigInt BigInt::operator<<(size_t n) const {
	return BigInt(std::string(isNeg ? "-" : "") + value + std::string(n, '0'));
}

CUDA_MEMBER BigInt BigInt::operator>>(size_t n) const {
	//if (n >= value.length())
	//	return 0;
	BigInt q = *this;
	std::string qs;
	while(q > 0)
	{
		if (q % 2 == 0)
			qs += "0";
		else
			qs += "1";
		q /= 2;
		// 111101001011111111100000110
	}
	// Invert string
	//std::string qsi;
	qs = qs.substr(1, qs.length() -1 );
	BigInt result(0);
	long int i = 0;
	while (i < qs.length())
	{
		//std::cout << BigInt(2).pow(i) * BigInt(qs[i]) << std::endl;
		//std::cout <<  BigInt(qs[i]) << std::endl;
		if (qs[i] == '1')
			result += BigInt(2).pow(i) * BigInt(1);
		i += 1;
	}
	//std::cout << result << std::endl;
	// for (int i = 0; i < qs.length(); i++)
	// 	qsi += qs[qs.length() - i - 1];
	// //std::cout << qsi  << std::endl;
	// return BigInt(qsi);
	return result;
	//return BigInt(std::string(isNeg ? "-" : "") + value.substr(0, value.length() - n));
}

// краткая запись сложения
CUDA_MEMBER BigInt &BigInt::operator+=(const BigInt &bigInt) {
	return *this = *this + bigInt;
}

// краткая запись вычитания
CUDA_MEMBER BigInt &BigInt::operator-=(const BigInt &bigInt) {
	return *this = *this - bigInt;
}

// краткая запись умножения
CUDA_MEMBER BigInt &BigInt::operator*=(const BigInt &bigInt) {
	return *this = *this * bigInt;
}

// краткая запись деления
CUDA_MEMBER BigInt &BigInt::operator/=(const BigInt &bigInt) {
	return *this = *this / bigInt;
}

// краткая запись взятия остатка
CUDA_MEMBER BigInt &BigInt::operator%=(const BigInt &bigInt) {
	return *this = *this % bigInt;
}

// краткая запись свдига влево
CUDA_MEMBER BigInt &BigInt::operator<<=(size_t n) {
	return *this = *this << n;
}

// краткая запись свдига вправо
CUDA_MEMBER BigInt &BigInt::operator>>=(size_t n) {
	return *this = *this >> n;
}

// префиксная запись инкремента
CUDA_MEMBER BigInt &BigInt::operator++() {
	return *this = *this + 1;
}

// префиксная запись декремента
CUDA_MEMBER BigInt &BigInt::operator--() {
	return *this = *this - 1;
}

// постфиксная запись инкремента
CUDA_MEMBER BigInt BigInt::operator++(int) {
	BigInt res = *this;
	*this = *this + BigInt(1);	

	return res;
}

// постфиксная запись декремента
CUDA_MEMBER BigInt BigInt::operator--(int) {
	BigInt res = *this;
	*this = *this - BigInt(1);

	return res;
}

// вывод числа в выходной поток
CUDA_MEMBER std::ostream &operator<<(std::ostream &stream, const BigInt &bigInt) {
	if (bigInt.getIsNeg())
		stream << "-";

	return stream << bigInt.getValue();
}

CUDA_MEMBER bool BigInt::operator &(int a)
{
	BigInt q = *this;
	std::string qs;
	while(q > 0)
	{
		if (q % 2 == 0)
			qs += "0";
		else
			qs += "1";
		q /= 2;
	}
	if (qs[0] == '1')
		return true;
	else
		return false;
}


// ввод числа из входного потока
CUDA_MEMBER std::istream &operator>>(std::istream &stream, BigInt &bigInt) {
	std::string value;
	stream >> value;
	bigInt = BigInt(value);

	return stream;
}


// CUDA_MEMBER BigInt &operator>>(BigInt &a, int b) {
// 	std::string binary = "";	
// 	int i = 0;
// 	while( i < a.value.length() )
// 	{
// 		switch ( a.value[i] )
// 		{
// 			case '0':
// 				binary += "0000";
// 				break;
// 			case '1':
// 				binary += "0001";
// 				break;
// 			case '2':
// 				binary += "0010";
// 				break;
// 			case '3':
// 				binary += "0011";
// 				break;
	// 		case '4':
	// 			binary += "0100";
	// 			break;
	// 		case '5':
	// 			binary += "0101";
	// 			break;
	// 		case '6':
	// 			binary += "0110";
	// 			break;
	// 		case '7':
	// 			binary += "0111";
	// 			break;
	// 		case '8':
	// 			binary += "1000";
	// 			break;
	// 		case '9':
	// 			binary += "1001";
	// 			break;
	// 	}
	// 	i += 1;
	// }	
	// std::bitset <128> bits(binary);
	// std::cout << bits.to_string() << std::endl;
	// return bits[127];

//}
